#include "hip/hip_runtime.h"
﻿/* 
CUDA-Programm zur Veranschaulichung der Herauforderungen des Profilings im Kontext der GPGPU-Programmierung
Bachelorarbeit Informatik, Universität Trier, 2024
Fabian Vecellio del Monego, 2024
*/

// ###################################################################################################################

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <sstream>
#include <chrono>
#include <string>
#include <cudaProfiler.h>
#include <hip/hip_runtime_api.h>

using int64 = long long int;  // weil "long" unter Windows nur 32 Bit hat und "long long int" unübersichtlich ist
using int32u = unsigned int;

// CUDA-Kernel #######################################################################################################

// Trivialer Primzahl-Test
__global__ void primeTest1(bool* array, int64 l, int64 u) {
    int64 index = threadIdx.x + blockIdx.x * blockDim.x; // Index der Zahl finden, die der Thread bearbeitet

    if (index >= l && index <= u) { // Range "durchlaufen"
        if (index > 1) {
            for (long i = 2; i <= sqrtf(index); i++) { // Suche nach Teiler bis zur Quadratwurzel der Zahl
                if (index % i == 0) {  // wenn die Zahl durch eine andere Zahl teilbar ist, ist sie keine Primzahl
                    array[index - l] = false;
                }
            }
        }
        else array[index - l] = false; // 0 und 1 sind keine Primzahlen
    }
    return;
}

// Performanterer Test
__global__ void primeTest2(bool* array, int64 l, int64 u) { 
    int64 index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index >= l && index <= u) {
        if (index > 2 && index % 2 != 0) { // Nur ungerade Zahlen prüfen
            for (long i = 3; i <= sqrtf(index); i += 2) { // Nur ungerade Teiler prüfen
                if (index % i == 0) {
                    array[index - l] = false;
                }
            }
        }
        else if (index == 2) {
            array[index - l] = true; // 2 ist eine Primzahl
        }
        else array[index - l] = false; // 0, 1 und gerade Zahlen sind keine Primzahlen
    }
    return;
}

// Noch performanterer Test
__global__ void primeTest3(bool* array, int64 l, int64 u) {
    int64 index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index == 2 || index == 3 || index > u) { // 2 und 3 sind Primzahlen
        return;
    }
    if (index <= 1 || index % 2 == 0 || index % 3 == 0) { // 0, 1 und Vielfache von 2 und 3 sind keine Primzahlen
        array[index - l] = false;
        return;
    }
    for (long i = 5; i <= sqrtf(index); i += 6) { // Primzahlen größer 3 folgen dem Muster 6k ± 1
        if (index % i == 0 || index % (i + 2) == 0)
            array[index - l] = false;
    }
    return;
}

// Noch performanterer Test, unsigned int, bis 4 Mrd.
__global__ void primeTest3Unsigned(bool* array, int32u u) { 
    int32u index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index == 2 || index == 3 || index > u) { // 2 und 3 sind Primzahlen
        return;
    }
    if (index <= 1 || index % 2 == 0 || index % 3 == 0) { // 0, 1 und Vielfache von 2 und 3 sind keine Primzahlen
        array[index] = false;
        return;
    }
    for (int32u i = 5; i <= sqrtf(index); i += 6) { // Primzahlen größer 3 folgen dem Muster 6k ± 1
        if (index % i == 0 || index % (i + 2) == 0)
            array[index] = false;
    }
    return;
}

// Noch performanterer Test, unsigned int, nur ungerade Zahlen
__global__ void primeTest3UnsignedOdd(bool* array, int32u u) {
    int32u index = threadIdx.x + blockIdx.x * blockDim.x;
    int32u i = index * 2 - 1; // Index anpassen gemäß 2i - 1, sodass i immer ungerade ist

    if (i == 1 || (i != 3 && i % 3 == 0)) { // 1 und Vielfache von 3 sind keine Primzahlen
        array[index] = false;
        return;
    }
    for (int32u j = 5; j <= sqrtf(i); j += 6) { // Primzahlen größer 3 folgen dem Muster 6k ± 1
        if (i % j == 0 || i % (j + 2) == 0) {
            array[index] = false;
            return;
        }
    }
    return;
}

// Noch performanterer Test, unsigned int, nur ungerade Zahlen
__global__ void primeTest3Inverted(bool* array, int32u u) {
    int32u index = threadIdx.x + blockIdx.x * blockDim.x;
    int32u i = index * 2 - 1; // Index anpassen gemäß 2i - 1, sodass i immer ungerade ist

    if (i == 1 || (i != 3 && i % 3 == 0)) { // 1 und Vielfache von 3 sind keine Primzahlen
        return;
    }
    for (int32u j = 5; j * j <= i; j += 6) { // Primzahlen größer 3 folgen dem Muster 6k ± 1
        if (i % j == 0 || i % (j + 2) == 0) {
            return;
        }
    }
    array[index] = false;
    return;
}

// Sieb des Eratosthenes
__global__ void sieveEratosthenes(bool* array, int64 u) { 
    int64 index = threadIdx.x + blockIdx.x * blockDim.x;

    if (array[index]) { // nur berechnen, wenn die Zahl noch nicht ausgeschlossen wurde
        if (index >= 2) {
            int64 i = index * index; // Noch nicht gefundene Primzahlen können nur größer als index^2 sein

            for (i; i <= u; i += index) { // Alle Vielfache als nicht prim markieren
                array[i] = false;
            }
        }
        else if (index == 0 || index == 1) { // 0 und 1 sind keine Primzahlen
            array[index] = false;
        }
    }
    return;
}

// Sieb des Eratosthenes, unsigned int, bis 4 Mrd.
__global__ void sieveEratosthenesUnsigned(bool* array, int32u u) {
    int32u index = threadIdx.x + blockIdx.x * blockDim.x;

    if (array[index]) { // nur berechnen, wenn die Zahl noch nicht ausgeschlossen wurde
        if (index >= 2 && index <= sqrtf(u)) { // Test, um nicht über 32 Bit hinaus zu rechnen
            int32u i = index * index; // Noch nicht gefundene Primzahlen können nur größer als index^2 sein

            for (i; i <= u; i += index) { // Alle Vielfache als nicht prim markieren
                array[i] = false;
            }
        }
        else if (index == 0 || index == 1) { // 0 und 1 sind keine Primzahlen
            array[index] = false;
        }
    }
    return;
}

__global__ void sieveEratosthenesUnsignedOdd(bool* array, int32u u) {
    int32u index = threadIdx.x + blockIdx.x * blockDim.x;

    // Nur ungerade Zahlen werden berücksichtigt
    int32u i = 2 * index + 1;

    if (index > 0 && i * i <= u * 2) {
        if (array[index]) {
            for (int32u j = i * i; j <= u * 2; j += i * 2) {
                array[(j / 2) + 1] = false;
            }
        }
    }
    return;
    /* Leider funktioniert dieser Kernel nicht korrekt.Im Bereich bis bspw. 1000 haut es noch hin, aber bei größeren 
       Intervallen werden nicht mehr alle Primzahlen gefunden. Desto größer das Intervall, desto mehr Primzahlen 
       bleiben unerkannt. Ich habe einige Zeit in das Debugging dieses Kernels gesteck, mich aufgrund der eigentlichen 
       Zielsetzugn dieser Arbeit und der ohnehin performanteren Methode primeTest3UnsignedOdd dagegen entschieden, 
       weiter Zeit in diese eigentlich sehr interessante Variation des Siebes des Eratosthenes zu stecken. Ich habe 
       den Kernel aber dennoch im Code belassen - vielleicht finde ich ja später einmal eine Lösung. */
}

// Sieb des Sundaram
__global__ void sieveSundaram(bool* array, int64 u) {
    int64 index = threadIdx.x + blockIdx.x * blockDim.x + 1; // Sieb funktioniert nicht mit 0, daher + 1

    if (index < u) { // nur berechnen, wenn die Zahl in der Range liegt
        int64 n = (u - 2) / 2; // obere Grenze anpassen

        for (int64 i = index; (index + i + 2 * index * i) <= n; i++) { // Iterieren bis (Obergrenze - 2) / 2
            array[index + i + 2 * index * i] = false; // Markieren der Zahlen i, sodass 2 * i + 1 != prim
        }
    }
    return;
}

// Test-Kernel
__global__ void primeTestDebug(bool* array, int32u n) {
    int64 i = threadIdx.x + blockIdx.x * blockDim.x + 1;

    for (int64 j = i; (i + j + 2 * i * j) <= n; j++) {
        array[i + j + 2 * i * j] = false;
    }
    return;
}

// Hilfsfunktionen ###################################################################################################

void clearInputBuffer() { // Hilfsfunktion, um den Eingabepuffer zu leeren
    int c;
    while ((c = getchar()) != '\n' && c != EOF) {}
    return;
}

void askMethod(int& method, std::string& kernelString) { // Funktion, um die Methode zu wählen
    int option;
    printf("Please choose a method. (1, 2, 3, 4, 5)\n");
    printf("1 = Trivial Test; 2 = Enhanced Test; 3 = Further enhanced Test; "
        "4 = Sieve of Eratosthenes; 5 = Sieve of Sundaram\n");
    printf("Enter method option: ");
    clearInputBuffer;
    scanf_s("%d", &option);

    switch (option) {
        case 0:     { method = option; kernelString = "DEBUG KERNEL";                 break; }
        case 1:     { method = option; kernelString = "primeTest1";                   break; }
        case 2:     { method = option; kernelString = "primeTest2";                   break; }
        case 3:     { method = option; kernelString = "primeTest3";                   break; }
        case 4:     { method = option; kernelString = "sieveEratosthenes";            break; }
        case 5:     { method = option; kernelString = "sieveSundaram";                break; }
        case 33:    { method = option; kernelString = "primeTest3Unsigned";           break; }
        case 44:    { method = option; kernelString = "sieveEratosthenesUnsigned";    break; }
        case 333:   { method = option; kernelString = "primeTest3UnsignedOdd";        break; }
        case 444:   { method = option; kernelString = "sieveEratosthenesUnsignedOdd"; break; }
        case 3333:  { method = option; kernelString = "primeTest3Inverted";           break; }
        default: {
            printf("Invalid option. Will commence with the default method: Trivial Test.\n");
            method = 1; // Standardfall ist der triviale Test
            kernelString = "primeTest1";
            break;
        }
    }

    printf("\n");
    return;
}

void askRange(int64& l, int64& u, int method) { // Funktion, um das zu prüfende Intervall zu wählen
    int option;
    printf("Please choose an upper bound. (1, 2, 3, 4, 5)\n");
    if (method <= 3) { // nur diese Methode erlauben l > 0
        printf("1 = 1,000; 2 = 100,000,000; 3 = 1,000,000,000; 4 = 4,000,000,000; 5 = custom range\n");
    }
    else {
        printf("1 = 1,000; 2 = 100,000,000; 3 = 1,000,000,000; 4 = 4,000,000,000; 5 = custom upper bound\n");
    }
    printf("Enter range option: ");
    clearInputBuffer;
    scanf_s("%d", &option);

    l = 0; // Standardwert für untere Grenze ist 0
    switch (option) {
        // Test-Option
        case 0: {
            u = 10;
            break;
        }
        // Reguläre Optionen
        case 1: {
            u = 1000;
            break;
        }
        case 2: {
            u = 100000000; // 100 Mio.
            break;
        }
        case 3: {
            u = 1000000000; // 1 Mrd.
            break;
        }
        case 4: {
            u = 4000000000; // 4 Mrd.
            break;
        }
        case 5: {
            printf("\nPlease define the range of natural numbers you want to check for primes.\n");
            if (method <= 3 || method == 33) { // nur diese Methode erlauben l > 0
                printf("Enter lower bound: ");
                clearInputBuffer;
                scanf_s("%lld", &l);
            }
            printf("Enter upper bound: ");
            clearInputBuffer;
            scanf_s("%lld", &u);
            if (l > u) { // wenn die obere Grenze  kleiner als die untere ist
                printf("Invalid range. Will commence with default bound: 50000000.\n");
                l = 0, u = 1000000;
            }
            if (l < 0) { // wenn die untere Grenze negativ ist
                l = 0;
                if (u < l) u = l; // wenn die obere Grenze jetzt kleiner als die untere ist
                printf("Adjusting range to natural numbers, since prime numbers are positive by definition.\n"
                    "New Range: %lld to %lld.\n", l, u);
            }
            if ((method == 33 || method == 333 || method == 3333 || 
                method == 44 || method == 444) && u >= 4294311961) {
                u = 4294311960; // maximaler Wert, bei dem die Schleife in primeTest3333 terminieren kann
                printf("This method can't handle numbers that big.\n"
                    "New Range: %lld to %lld.\n", l, u);
		    }
            break;
        }
        // Test-Optionen
        case 6: {
            u = 10000000; // 10 Mio.
            break;
        }
        case 7: {
            u = 2000000000; // 2 Mrd.
            break;
        }
        case 8: {
            u = 8000000000; // 8 Mrd.
            break;
        }
        case 9: {
            l = 10;
            u = 1000;
            break;
        }
        default: {
            u = 1000000; // Standardwert: 1 Million
            printf("Invalid option. Will commence with the default bound: %lld.\n", u);
        }
    }
    printf("\n");
    return;
}

void askBlockSize(int& blockSize) { // Funktion, um die Blockgröße zu wählen oder alle zu testen
    char input[10];
    printf("Please choose the amount of threads per block. (32, 64, 128, 256, 512, 1024, all)\n");
    printf("Enter block size: ");
    clearInputBuffer();
    fgets(input, sizeof(input), stdin);

    input[strcspn(input, "\n")] = 0; // Eingabe bereinigen

    if (strcmp(input, "all") == 0) {
        blockSize = 1;
        return;
    }

    bool isValidNumber = true; // Prüfen, ob die Eingabe eine Zahl ist
    for (int i = 0; input[i] != '\0'; i++) {
        if (!isdigit(input[i])) {
            isValidNumber = false;
            break;
        }
    }

    if (isValidNumber) {
        int option = atoi(input);
        if (option % 32 == 0 && option <= 1024) {
            blockSize = option;
            return;
        }

        if (option >= 1 && option <= 6) {
            switch (option) {
                case 1: { blockSize = 32;   return; }
                case 2: { blockSize = 64;   return; }
                case 3: { blockSize = 128;  return; }
                case 4: { blockSize = 256;  return; }
                case 5: { blockSize = 512;  return; }
                case 6: { blockSize = 1024; return; }
            }
            return;
        }
    }

    printf("Invalid option. Will commence with the default block size of 256.\n");
    blockSize = 256; // Standardfall
    return;
}

void output(bool* array, int64 l, int64 u, int64 rangeSize, long primeCount) { // Ausgabe-Funktion
    if (primeCount > 0) {
        std::string input = "y"; // Ausgabe der gefunden Primzahlen ist standardmäßig aktiviert

        printf("\n");
        if (primeCount > 25) { // keine automatische Ausgabe, wenn mehr als x Primzahlen gefunden wurden
            printf("Dou you want a list? (y/n)\n");
            printf("Enter y or n: ");
            std::getline(std::cin, input);
        }

        if (input == "y" || input == "Y") {
            // printf("\nPrimes in range %d to %d:\n", l, u);
            int lineLength = 0;
            std::ostringstream oss;
            std::string primeString;
            printf("\n");
            for (int64 i = 0; i < rangeSize; i++) { // Ausgabe der Primzahlen
                if (array[i]) { // wenn die Zahl eine Primzahl ist
                    oss.str("");
                    oss << i;
                    primeString = oss.str();
                    if (lineLength + primeString.length() + 1 > 120) { // Prüfen, 
                        // ob das Hinzufügen der nächsten Zahl die maximale Zeilenlänge überschreitet
                        std::cout << std::endl;  // Zeilenumbruch
                        lineLength = 0;   // Zähler zurücksetzen
                    }
                    std::cout << primeString << " ";
                    lineLength += primeString.length() + 1; // Aktualisieren der aktuellen Zeilenlänge
                }
            }
            printf("\n");
        }
        else {
            printf("\nOkay, bye!\n");
        }
    }
    return;
}

// Hauptfunktion #####################################################################################################

int main() {
    // Methode wählen
    int method;
    std::string kernelString;
    askMethod(method, kernelString);
    bool needsOddTransformation = false; // ob nur ungerade Zahlen geprüft werden
    bool needsSundaramTransformation = false; // ob das Array transformiert werden muss
    bool needsInversion = false; // ob das Array invertieert werden muss

    // Welche Zahlen sollen geprüft werden?
    int64 l, u; // untere und obere Grenze
    askRange(l, u, method);
    int64 rangeSize = u + 1 - l;
    int64 arraySize = rangeSize;
    int32u n = (u - 2) / 2; // für Sundaram-Test-Kernel

    if (method >= 100) { // für Methoden, die nur ungerade Zahlen prüfen
        arraySize = rangeSize / 2; // Array-Größe anpassen
        needsOddTransformation = true;
    }

    // Parameter für den Kernel setzen
    int blockSize; // Anzahl der Threads pro Block
    askBlockSize(blockSize);

    bool testAll = false;
    double times[6]; // Zeiten für die verschiedenen Blockgrößen
    if (blockSize == 1) { // alle Blockgrößen testen
        testAll = true;
        blockSize = 32;
	}

    // ==============================================================================================================

    printf("\nInitializing arrays ... ");

    // Array auf dem Host anlegen und initialisieren
    bool* array;
    array = (bool*)malloc(arraySize * sizeof(bool)); // Array auf dem Host anlegen
    for (int64 i = 0; i < arraySize; i++) { // Array mit true initialisieren
        array[i] = true;
    }

    for (int run = 0; run < 6; run++) { // Schleife, wenn alle Blockgrößen getestet werden

        // Array auf dem Device anlegen und kopieren
        bool* deviceArray;
        hipError_t error; // Fehler-Variable
        error = hipMalloc((void**)&deviceArray, arraySize * sizeof(bool));
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA error in hipMalloc: %s\n", hipGetErrorString(error));
        }
        error = hipMemcpy(deviceArray, array, arraySize * sizeof(bool), hipMemcpyHostToDevice);
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA error in hipMemcpy (HostToDevice): %s\n", hipGetErrorString(error));
        }

        // Kernel-Konfiguration berechnen
        int blocks = (arraySize + blockSize - 1) / blockSize; // Anzahl der Blöcke berechnen,
        // sodass immer genug Threads für alle Zahlen vorhanden sind (in 256er Schritten)

        printf("\nLaunching CUDA kernel %s with %d threads per block ... ", kernelString.c_str(), blockSize);
        if (!testAll) printf("\nSearching for primes in range %lld to %lld ... ", l, u);
        hipProfilerStart;

        // Zeitmessung starten
        auto start = std::chrono::high_resolution_clock::now();

        // Kernel starten
        switch (method) {
            case 0: { // Test-Kernel 
                primeTestDebug <<< blocks, blockSize >>> (deviceArray, n);
                needsSundaramTransformation = true;
                break;
            }
            case 1: { // Trivialer Test
                primeTest1 <<< blocks, blockSize >>> (deviceArray, l, u);
                break;
            }
            case 2: { // Verbesserter Test
                primeTest2 <<< blocks, blockSize >>> (deviceArray, l, u);
                break;
            }
            case 3: { // Weiter verbesserter Test
                primeTest3 <<< blocks, blockSize >>> (deviceArray, l, u);
                break;
            }
            case 4: { // Sieb des Eratosthenes
                sieveEratosthenes <<< sqrtf(blocks), blockSize >>> (deviceArray, u);
                break;
            }
            case 5: { // Sieb des Sundaram
                sieveSundaram <<< sqrtf(blocks), blockSize >>> (deviceArray, u);
                needsSundaramTransformation = true;
                break;
            }
            case 33: { // Weiter verbesserter Test mit unsigned int (32 Bit)
                int32u uTemp = u;
                primeTest3Unsigned <<< blocks, blockSize >>> (deviceArray, uTemp);
                break;
            }
            case 44: { // Sieb des Eratosthenes mit unsigned int (32 Bit)
                int32u uTemp = u;
                sieveEratosthenesUnsigned <<< sqrtf(blocks), blockSize >>> (deviceArray, uTemp);
                break;
            }
            case 333: { // Weiter verbesserter Test mit unsigned int (32 Bit), nur ungerade Zahlen
                int32u uTemp = u / 2;
                primeTest3UnsignedOdd <<< blocks, blockSize >>> (deviceArray, uTemp);
                break;
            }
            case 444: { // Weiter optimiertes Sieben mit unsigned int (32 Bit), nur ungerade Zahlen
                int32u uTemp = u / 2;
                sieveEratosthenesUnsignedOdd <<< sqrtf(blocks), blockSize >>> (deviceArray, uTemp);
                break;
            }
            case 3333: { // Noch weiter verbesserter Test
                int32u uTemp = u / 2;
                primeTest3Inverted <<< blocks, blockSize >>> (deviceArray, uTemp);
                needsInversion = true;
                break;
            }
        }

        hipDeviceSynchronize(); // Warten, bis alle Threads fertig sind
        hipProfilerStop;

        // Zeitmessung stoppen
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        times[run] = elapsed.count();

        if (!testAll || run == 5) { // wenn nur eine Blockgröße getestet wird oder alle getestet wurden
            // Ergebnis von Device auf Host kopieren
            error = hipMemcpy(array, deviceArray, arraySize * sizeof(bool), hipMemcpyDeviceToHost);
            if (error != hipSuccess) {
                fprintf(stderr, "CUDA error in hipMemcpy (DeviceToHost): %s\n", hipGetErrorString(error));
            }

            run = 6; // Schleife beenden
        }
        else {
            blockSize *= 2; // Blockgröße verdoppeln
        }

        // Speicher wieder freigeben
        error = hipFree(deviceArray); 
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA error in hipFree: %s\n", hipGetErrorString(error));
        }
    }

    // Ausgabe vorbereiten ==========================================================================================

    if (needsInversion) { // falls Primzahlen als false markiert wurden
        printf("\nInverting array ... ");
        bool* tempArray = (bool*)malloc(rangeSize * sizeof(bool)); // Neues Array anlegen

        for (int64 i = 0; i <= u; i++) { // Array mit false initialisieren
            tempArray[i] = false;
        }

        for (int64 i = 0; i < arraySize; i++) {
            if (!array[i]) {
				tempArray[i] = true;
			}
        }

        array = tempArray; // Pointer umleiten
    }

    if (needsOddTransformation) { // für Methoden, die nur ungerade Zahlen prüfen
        printf("\nTransforming array ... ");
        bool* tempArray = (bool*)malloc(rangeSize * sizeof(bool)); // Neues Array anlegen

        for (int64 i = 0; i <= u; i++) { // Array mit false initialisieren
            tempArray[i] = false;
        }

        for (int64 i = 2; i < arraySize; i++) { // Array transformieren gemäß 2i - 1
            tempArray[2 * i - 1] = array[i];
        }

        tempArray[2] = true; // Sonderfall 2 kann bei Methode zu ungeraden Zahlen nicht berücksichtigt werden

        array = tempArray; // Pointer umleiten
    }

    if (needsSundaramTransformation) { // für das Sundaram-Sieb
        if (!needsOddTransformation) printf("\nTransforming array ... ");
        bool* tempArray = (bool*)malloc(rangeSize * sizeof(bool)); // Neues Array anlegen

        for (int64 i = 0; i <= u; i++) { // Array mit false initialisieren
            tempArray[i] = false;
        }

        for (int64 i = 0; i <= (u - 2) / 2; i++) { // Array transformieren
            if (array[i]) {
                int64 prime = 2 * i + 1;
                if (prime == 1) { // 1 ist keine Primzahl, 2 aber schone 
                    tempArray[prime + 1] = true;
                }
                else {
                    tempArray[prime] = true;
                }
            }
        }

        std::copy(tempArray, tempArray + rangeSize, array); // Array kopieren
        free(tempArray); // Speicher freigeben
    }

    // Anzahl der Primzahlen zählen und ausgeben ====================================================================
    long primeCount = 0;
    for (int64 i = 0; i < rangeSize; i++) {
        if (array[i]) { // Anzahl der Primzahlen zählen
            primeCount++;
        }
    }

    // Ausgabe-Methode instruieren 
    printf("\n\nFound %d primes in range %lld to %lld. ", primeCount, l, u);
    if (!testAll) {
        printf("This took %lf seconds.", times[0]);
    }
    else {
        printf("\n\n%10s %10s %10s %10s %10s %10s %10s", "Block size:", "32", "64", "128", "256", "512", "1024");
        printf("\n%10s %10.6lf %10.6lf %10.6lf %10.6lf %10.6lf %10.6lf \n", "Time (s):  ",
            times[0], times[1], times[2], times[3], times[4], times[5]);
    }
    output(array, l, u, rangeSize, primeCount); // Gibt Werte an Ausgabe-Funktion weiter
    
    free(array); // Speicher freigeben  

    return 0;
}

/* Ausgewählte Ergebnisse ###########################################################################################
    1.000.000.000: 
        1) 455s, 2) 234s, 3) 138s, 4) 98s, 5) 138s; 
        33) 33s, 44: 82s, 
        333) 17s, 444) 27s [falsch]
        3333) 12s, 
    4.000.000.000: 
        3) 1107s, 4) 400s,
        33) 270s, 44) 330s, 
        333) 134s 
        3333) 93s 
 */
